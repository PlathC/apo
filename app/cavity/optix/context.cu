#include <apo/core/logger.hpp>
#include <apo/gpu/setup.hpp>

#include "optix/context.cuh"

namespace apo::optix
{
    static void contextLogCallback( unsigned int level, const char * tag, const char * message, void * )
    {
        apo::logger::debug( "[OPTIX][{}][{}]: {}", level, tag, message );
    }

    Context::Context()
    {
        int numDevices;
        hipGetDeviceCount( &numDevices );
        logger::debug( "Found {} CUDA devices", numDevices );

        optixCheck( optixInit() );
        // for this sample, do everything on one device
        const int deviceID = 0;
        apo::gpu::cudaCheck( hipSetDevice( deviceID ) );
        apo::gpu::cudaCheck( hipStreamCreate( &m_stream ) );

        hipGetDeviceProperties( &m_deviceProps, deviceID );
        logger::debug( "Running on device: {}", m_deviceProps.name );

        hipCtx_t cuCtx = 0; // zero means take the current context
        optixCheck( optixInit() );
        OptixDeviceContextOptions options = {};
        options.logCallbackFunction       = &contextLogCallback;
        options.logCallbackLevel          = 4;
        options.validationMode            = OPTIX_DEVICE_CONTEXT_VALIDATION_MODE_ALL;
        optixCheck( optixDeviceContextCreate( cuCtx, &options, &m_optiXContext ) );
    }

    Context::Context( Context && other ) noexcept
    {
        std::swap( m_stream, other.m_stream );
        std::swap( m_deviceProps, other.m_deviceProps );
        std::swap( m_optiXContext, other.m_optiXContext );
    }

    Context & Context::operator=( Context && other ) noexcept
    {
        std::swap( m_stream, other.m_stream );
        std::swap( m_deviceProps, other.m_deviceProps );
        std::swap( m_optiXContext, other.m_optiXContext );

        return *this;
    }

    Context::~Context() = default;
} // namespace apo::optix
