#include "apo/gpu/benchmark.cuh"

namespace apo::gpu
{
    double timer_ms( const Benchmark::Task & task )
    {
        hipEvent_t start, stop;
        hipEventCreate( &start );
        hipEventCreate( &stop );

        hipEventRecord( start );

        task();

        hipEventRecord( stop );

        hipEventSynchronize( stop );
        float milliseconds = 0;
        hipEventElapsedTime( &milliseconds, start, stop );

        hipEventDestroy( start );
        hipEventDestroy( stop );
        return milliseconds;
    }
} // namespace apo::gpu
